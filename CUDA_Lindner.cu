#include "hip/hip_runtime.h"
#include <stdlib.h>
#include<stdio.h>
#include <iostream>
#include <string>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <gmp.h>
#include <mpfr.h>
#include <fplll.h>
#include "fplll/defs.h"
#include "fplll/util.h"
#include <fplll/nr/nr.h>
#include <fplll/nr/numvect.h>
using namespace std;
using namespace fplll;

#ifndef TESTDATADIR
#define TESTDATADIR ".."
#endif
#define N  1600
#define THREADS_PER_BLOCK 40
#ifndef CAFFE_COMMON_CUH_
#define CAFFE_COMMON_CUH_

#include <hip/hip_runtime.h>

  #if !defined(__CUDA_ARCH__) || __CUDA_ARCH__ >= 600

  #else
  static __inline__ __device__ double atomicAdd(double *address, double val) {
    unsigned long long int* address_as_ull = (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    if (val==0.0)
      return __longlong_as_double(old);
    do {
      assumed = old;
      old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val +__longlong_as_double(assumed)));
    } while (assumed != old);
    return __longlong_as_double(old);
  }


  #endif
#endif

/**
   @brief Read T from `input_filename`.
   @param X T (T is usually a ZZ_mat<ZT> or a vector<Z_NR<ZT>>
   @param input_filename
   @return zero if the file is correctly read, 1 otherwise.
*/

__host__
void read_file(ZZ_mat<mpz_t> &X, const char *input_filename) {
  int status = 0;
  ifstream is;
  is.exceptions(std::ifstream::failbit | std::ifstream::badbit);
  try {
    is.open(input_filename);
    is >> X;
    is.close();
  }
  catch (const ifstream::failure&) {
    status = 1;
    cerr << "Error by reading " << input_filename << "." << endl;
    cout << status << endl;
    cout << is.rdstate() << endl;
  }

}

void read_vector(vector<double> &vector, const char *filename) {
	ifstream is;
	string string;
	is.open(filename);
	while (!is.eof()){
		while (getline(is, string)){
		if (string.c_str() !=" ") {
				vector.push_back(atof(string.c_str()));
			}
		}
	}
	is.close();
}


/*int read_dimension (const char *input_filename) {
	int dim;
	string input;
	ifstream is;
	is.open(input_filename);
	try {
		getline(is, input);
		if (input.size() > 0) {
			dim = atoi(input.c_str());
			return dim;
		}
		return 0;
	}
	catch (const ifstream::failure&) {
		cerr << "Error reading " << input_filename << "." << endl;
		return 0;
	}
}*/

template <class T, class U> NumVect<FP_NR<T>> addRow (MatrixRow<Z_NR<U>> &&vector, FP_NR<T> num){
	
	NumVect<FP_NR<T>> result(vector.size());
	for (int i = 0; i < vector.size(); i++) {

		result[i].add(num, vector[i].get_ld(), MPFR_RNDN);// This is FP_NR class's function to implement multiplication,
														 // allowing us to avoid confusing mpfr ang mpz functions in case
														 // of usage of MPFR or GMP Libraries.
	}
	return result;

}

template <class T> NumVect<FP_NR<T>> addRow (MatrixRow<FP_NR<T>> &&row, FP_NR<T> num) {
	NumVect<FP_NR<T>> toReturn(row.size());
	for (int i = 0; i < row.size(); i++) {

		toReturn[i].add(row[i], num, MPFR_RNDN);// See comments of mult() function above
	}
	return toReturn;
}

template <class T, class U> FP_NR<T> dotProduct (MatrixRow<Z_NR<U>> &&vector1, NumVect<FP_NR<T>> &vector2, int length1, int length2) {
	
	FP_NR<T> sum = FP_NR<T> (0.0);
	NumVect<FP_NR<T>> vect(1);
	vect.fill(0.0);
	int i = max(length1, length2);
	for (int j = 0; j <= i - 1; j++) {
		vect[0].mul(vector2[j], vector1[j].get_ld(), MPFR_RNDN);
		sum = sum + vect[0];
		vect.fill(0.0);

	}
	return sum;

}

template <class T> FP_NR<T> dotProduct (NumVect<FP_NR<T>> &vector1, NumVect<FP_NR<T>> &vector2, int length1, int length2) {
	
	FP_NR<T> sum = FP_NR<T> (0.0);
	int i = max(length1, length2);
	for (int j = 0; j <= i - 1; j++) {

		sum = sum + vector1[j]*vector2[j];

	}
	return sum;

}

NumVect<NumVect<FP_NR<mpfr_t>>> gSO (ZZ_mat<mpz_t> & base, NumVect<NumVect<FP_NR<mpfr_t>>> & gramBase) {
	ZZ_mat<mpz_t> identity;//Identity matrix of integers, used in LLL
	ZZ_mat<mpz_t> idTrans;//Transposed version of identity
	int dimension = base.get_cols();//Retrieve dimension of lattice
	FP_NR<mpfr_t> l1 = 0.0;
	FP_NR<mpfr_t> l2 = 0.0;
	FP_NR<mpfr_t> l = 0.0;
	FP_NR<mpfr_t> zero = FP_NR<mpfr_t> (0.0);
	NumVect<FP_NR<mpfr_t>> vect(dimension);
	NumVect<FP_NR<mpfr_t>> muVect(dimension);
	muVect.fill(0.0);
	vect.fill(0.0);
	vect = addRow(base[0], zero);
	gramBase[0] = vect;
	for (int i = 1; i < dimension; i++) {
		vect = addRow(base[i], zero);
		for (int j = i - 1; j >= 0; j--) {
			l1 = dotProduct(base[i], gramBase[j], base[i].size(), gramBase[j].size());
			l2 = dotProduct(gramBase[j], gramBase[j], gramBase[j].size(),gramBase[j].size());
			l = l1 / l2;
			muVect.mul(gramBase[j], l);
			vect.sub(muVect);
		}
		gramBase[i] = vect;
	}
	return gramBase;// Return GSO-ed base

}

void get_gram(ZZ_mat<mpz_t> &base, FP_mat<mpfr_t> &gram) {
	int dimension = base.get_cols();
	NumVect<NumVect<FP_NR<mpfr_t>>> gramBase(dimension);
	gramBase = gSO(base, gramBase);
	for (int i = 0; i < dimension; i++) {
		for (int j = 0; j < dimension; j++) {
			gram[i][j] = gramBase[i][j];
		}
	}
}

void preprocess(ZZ_mat<mpz_t> &lattice, FP_mat<mpfr_t> &gram, vector<vector<double>> &doubleLattice,  vector<vector<double>> &doubleGram) {
	int rows = lattice.get_rows();
	int cols = lattice.get_cols();
	for (int i = 0; i < rows; i++) {
		doubleLattice[i].resize(cols);
		doubleGram[i].resize(cols);
		for (int j = 0; j < cols; j++) {
			doubleLattice[i][j] = lattice[i][j].get_si();
			doubleGram[i][j] = gram[i][j].get_d();		
		}
	}
}

//==========================================================================================================



__global__ void dotProduct(double *vector1, double *vector2, double *total, int dim) {
  __shared__ double temp[THREADS_PER_BLOCK];
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  if (index < dim) {
  	 temp[index] =  vector1[index] * vector2[index];
  }
  __syncthreads();
  if (0 == threadIdx.x) {
  		double sum = 0;
  		for (int i = 0; i < dim; i++) {
  			sum = sum + temp[i];
  		}
  	atomicAdd(total, sum);
  }
}

__global__ void vectorAdd (double *vector1, double *vector2, double *result) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	result[threadIdx.x] = vector1[index] + vector2[index];
}

__global__ void vectorSub (double *vector1, double *vector2, double *result) {
	int index = threadIdx.x +blockIdx.x * blockDim.x;
	result[threadIdx.x] = vector1[index] - vector2[index];
}    



double dot(vector<double> &vector1, vector<double> &vector2, int dim) {
	double * realVector1;
	double * realVector2;
	double * total;
	double * result = new double;
	hipMallocManaged (&realVector1, dim * sizeof(double));
	hipMallocManaged(&realVector2, dim * sizeof(double));
	hipMallocManaged(&total, 1 * sizeof(double));
	*total = 0.0;
	for (int i = 0; i < dim; i++) {
		realVector1[i] = vector1[i];
		realVector2[i] = vector2[i];
	}
	dotProduct<<<1,  dim>>> (realVector1, realVector2, total, dim);
	hipDeviceSynchronize(); 
	*result = *total;
	hipFree(realVector1);
	hipFree(realVector2);
	hipFree(total);
	return *result;
}

double integer_production (vector <double *> &list, vector<vector<double>> &gramBase, double* cuda_target, int index, int dim, int position) {
	double * c1;
	double * list_element;
	double * gram_temp;
	double * unrounded_integer;
	double   result = 0.0;
	hipMallocManaged (&c1, dim * sizeof(double));
	hipMallocManaged (&list_element, dim * sizeof(double));
	hipMallocManaged (&unrounded_integer, dim * sizeof(double));
	hipMallocManaged (&gram_temp, dim * sizeof(double));
	for (int i = 0; i < dim; i++) {
		list_element[i] = list[index][i];
		gram_temp[i] = gramBase[position][i];
		c1[i] = 0.0;
	}
	*unrounded_integer = 0.0;
	vectorSub<<<1, dim>>> (cuda_target, list_element, c1);
	hipDeviceSynchronize();
	dotProduct<<<1, dim>>> (c1, gram_temp, unrounded_integer, dim);
	hipDeviceSynchronize();
	result = *unrounded_integer;
	hipFree (list_element);
	hipFree(c1);
	hipFree(gram_temp);
	hipFree (unrounded_integer);
	return result;
}

void lindner (ZZ_mat<mpz_t> lattice, FP_mat<mpfr_t> gram, double* target, vector<int> buffer) {
	int dim = lattice.get_rows();
	vector<vector<double>> base (dim);
	vector<vector<double>> gramBase (dim);
	vector<double *> list (dim);
	list[0] = new double [dim];
	int k = 1;
	double result = 0.0;
	preprocess(lattice, gram, base, gramBase);
	double *gramDP = new double[dim];
	double *cuda_target;
	hipMallocManaged(&cuda_target, dim * sizeof(double));
	lattice.clear();
	gram.clear();
	for (int i = 0; i < dim; i++) {
		cuda_target[i] = target[i];
		list[0][i] = 0.0;
		gramDP[i] = dot(gramBase[i], gramBase[i], dim);
	}
	delete[] target;
	for (int i = dim -1; i >= 0; i--) {
		double ** tempList = new double*[dim];
		for (int j = 0; j < k; j++) {
			result = integer_production(list, gramBase, cuda_target, j, dim, i);
			cout << result << endl;
		}
		delete[] tempList;
	}
}



int main(int argc, char** argv) {
	//int dim = atoi(argv[1]);
	int dim = 5;
	ZZ_mat<mpz_t> lattice;
	FP_mat<mpfr_t> gramLattice;
	vector<double> target;
	vector<int> buffer (dim, 2);
	if (dim > 0) {
		lattice.resize(dim, dim);
		read_file(lattice, "storage");
		gramLattice.resize(dim, dim);
		get_gram(lattice, gramLattice);
		read_vector(target, "vector");
	}
	else {
		cout << "No dimension entered, program will exit." << endl;
	}
	double *targetReal = new double [dim];
	for (int i = 0; i < dim; i++) {
		targetReal[i] = target[i];
	}
	cout << endl;
	lindner (lattice, gramLattice, targetReal, buffer);
	return 0;

}